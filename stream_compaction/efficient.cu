#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void upSweep(int N, int offset, int* iData) {
            int offDown1 = offset >> 1;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            index *= offset;
            index += offDown1 - 1;
            if (index + offDown1 >= N) {
                return;
            }
            iData[index + offDown1] = iData[index] + iData[index + offDown1];

        }

        __global__ void downSweep(int N, int offset, int* iData) {
            int offDown1 = offset >> 1;
            int index = blockIdx.x * blockDim.x + threadIdx.x;

            index *= offset;
            index += offDown1 - 1;
            if (index + offDown1 >= N) {
                return;
            }
            //printf("index = %d, offDown1 = %d, value = %d \n", index, offDown1, iData[index] + iData[index + offDown1]);
            //printf(" |index left: %d, value left: %d, index right: %d, value right: %d, new value = %d | \n", index, iData[index], index + offDown1, iData[index + offDown1], iData[index] + iData[index + offDown1]);
            int temp = iData[index + offDown1];
            iData[index + offDown1] = iData[index] + iData[index + offDown1];
            iData[index] = temp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            int* dev_idata;
            int pow2N = ilog2ceil(n);
            int size = 1 << pow2N;
            int* padded = new int[size];
            for (int i = 0; i < size; i++) {
                if (i >= n || i == 0) {
                    padded[i] = 0;
                }
                else {
                    padded[i] = idata[i];
                }
            }

            hipMalloc((void**)&dev_idata, size * sizeof(int));
            hipMemcpy(dev_idata, padded, size * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 64;
            int gridSize = (size + blockSize - 1) / blockSize;
            int offset = 2;
            for (int d = 0; d < pow2N; d++) {
                offset = 1 << d + 1;
                upSweep << <gridSize, blockSize >> > (size, offset, dev_idata);
            }

            hipMemset(dev_idata + size - 1, 0, sizeof(int));

            for (int d = 0; d < pow2N; d++) { 
                downSweep << <gridSize, blockSize >> > (size, offset, dev_idata);
                offset = offset >> 1;
            }

            hipMemcpy(odata, dev_idata, size * sizeof(int), hipMemcpyDeviceToHost);

            for (int i = 1; i < size; i++) {
                odata[i - 1] = odata[i];
            }

            hipFree(dev_idata);
            timer().endGpuTimer();
        }

        __global__ void kernResetIntBuffer(int N, int* intBuffer, int value) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index < N) {
                intBuffer[index] = value;
            }
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            printf("n = %d", n);
            int pow2N = ilog2ceil(n);
            int size = 1 << pow2N;
            int* padded = new int[size];
            for (int i = 0; i < size; i++) {
                if (i >= n) {
                    padded[i] = 0;
                }
                else {
                    padded[i] = idata[i];
                }
            }

            int* dev_idata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int* dev_bool;
            int* host_bool = new int[n];
            int* host_scanResult = new int[n];
            int* dev_indices;
            int* dev_scatter;

            hipMalloc((void**)&dev_bool, n * sizeof(int));
            hipMalloc((void**)&dev_indices, n * sizeof(int));
            hipMalloc((void**)&dev_scatter, n * sizeof(int));

            int blockSize = 64;
            int gridSize = (n + blockSize - 1) / blockSize;

            Common::kernMapToBoolean << <gridSize, blockSize >> > (n, dev_bool, dev_idata);
            hipMemcpy(host_bool, dev_bool, n * sizeof(int), hipMemcpyDeviceToHost);
            timer().endGpuTimer();
            scan(n, host_scanResult, host_bool);
            timer().startGpuTimer();

            hipMemcpy(dev_indices, host_scanResult, n * sizeof(int), hipMemcpyHostToDevice);
            kernResetIntBuffer << <gridSize, blockSize >> > (n, dev_scatter, 0);
            Common::kernScatter << <gridSize, blockSize >> > (n, dev_scatter, dev_idata, dev_bool, dev_indices);

            for (int i = 0; i < n; i++) {
                printf("bool[%d] = %d : ", i, host_bool[i]);
            }
            printf("\n");
            
            for (int i = 0; i < n; i++) {
               printf("scan[%d] = %d : ", i, host_scanResult[i]);
            }
            printf("\n");

            hipMemcpy(odata, dev_scatter, n * sizeof(int), hipMemcpyDeviceToHost);
            int returnNum = 0;
            for (int i = 0; i < n; i++) {
                printf("result[%d] = %d : ", i, odata[i]);
                if (odata[i] == 0) { break; }
                returnNum++;
            }
            printf("\n");


            hipFree(dev_idata);
            hipFree(dev_bool);
            hipFree(dev_scatter);

            timer().endGpuTimer();
            return returnNum;
        }
    }
}
